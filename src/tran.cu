#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <random>
#include <tensor.hpp>

using namespace nn::global;

class RandomGenerator {
	std::mt19937 gen;

  public:
	RandomGenerator() : gen(std::random_device{}()) {}
	int getInt(int start, int end) {
		if (start > end)
			return start;
		std::uniform_int_distribution<> dist(start, end);
		return dist(gen);
	}

	std::mt19937 &getEngine() { return gen; }
};

static RandomGenerator rng;
int getAction(const int start, const int end) {
	return rng.getInt(start, end);
}

int getBiasedAction(int min_val, int max_val) {
	if (min_val == max_val)
		return min_val;

	// static constexpr float edge_bias_prob = 0.2f; // more bias toward edges
	float p = getAction(0, 10000) / 10000.0f;

	// if (p < edge_bias_prob) {
	//     // Pick left/top edge or right/bottom edge
	//     if (getAction(0, 1) == 0) {
	//         // Strong push toward min side
	//         return min_val;
	//     } else {
	//         // Strong push toward max side
	//         return max_val;
	//     }
	// } else {
	// Occasionally choose something in between
	return getAction(min_val, max_val);
	// }
}

constexpr int GRID_DIM = 28;
constexpr size_t SIZE = GRID_DIM * GRID_DIM; // 784

struct BoundingBox {
	int x_min = GRID_DIM;
	int y_min = GRID_DIM;
	int x_max = -1;
	int y_max = -1;
};

__global__ void moveKernel(const ValueType *original_data, ValueType *new_data, int horizontal_shift, int vertical_shift) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= SIZE)
		return;

	int dest_row = idx / GRID_DIM;
	int dest_col = idx % GRID_DIM;

	int source_row = dest_row - vertical_shift;
	int source_col = dest_col - horizontal_shift;

	if (source_row >= 0 && source_row < GRID_DIM &&
	    source_col >= 0 && source_col < GRID_DIM) {
		int source_idx = source_row * GRID_DIM + source_col;
		new_data[idx] = original_data[source_idx];
	}
}

// Simple device RNG for noise
__device__ float simpleHashRand(int seed, int idx) {
	unsigned int x = seed ^ idx;
	x ^= x << 13;
	x ^= x >> 17;
	x ^= x << 5;
	return (x % 1000) / 1000.0f; // [0,1)
}

// Kernel to add noise in range [noise_range_low, noise_range_high]
__global__ void addNoiseKernel(ValueType *data, float noise_range_low, float noise_range_high, int seed) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= SIZE)
		return;

	if (data[idx] > 0) {
		float noise = noise_range_low + (noise_range_high - noise_range_low) * simpleHashRand(seed, idx);
		float val = data[idx] + noise;

		val = fminf(fmaxf(val, 0.0f), 1.0f); // clamp to [0,1]

		data[idx] = val;
	}
}

// Updated move function
void move(const nn::global::Tensor &p, nn::global::Tensor &result) {
	std::vector<ValueType> host_data(SIZE);
	hipMemcpy(host_data.data(), p.getGpuData(), SIZE * sizeof(ValueType), hipMemcpyDeviceToHost);

	BoundingBox box;
	bool content_found = false;
	for (int i = 0; i < SIZE; ++i) {
		if (host_data[i] > 0.05) {
			content_found = true;
			int row = i / GRID_DIM;
			int col = i % GRID_DIM;
			box.x_min = std::min(box.x_min, col);
			box.x_max = std::max(box.x_max, col);
			box.y_min = std::min(box.y_min, row);
			box.y_max = std::max(box.y_max, row);
		}
	}

	if (!content_found) {
		hipMemset(result.getGpuData(), 0, SIZE * sizeof(ValueType));
		return;
	}

	int h_shift_min = -box.x_min;
	int h_shift_max = (GRID_DIM - 1) - box.x_max;

	int v_shift_min = -box.y_min;
	int v_shift_max = (GRID_DIM - 1) - box.y_max;

	int final_h_shift = getBiasedAction(h_shift_min, h_shift_max);
	int final_v_shift = getBiasedAction(v_shift_min, v_shift_max);

	std::size_t blockSize = 256;
	std::size_t numBlocks = (SIZE + blockSize - 1) / blockSize;

	hipMemset(result.getGpuData(), 0, SIZE * sizeof(ValueType));

	moveKernel<<<numBlocks, blockSize>>>(
	    p.getGpuData(),
	    result.getGpuData(),
	    final_h_shift,
	    final_v_shift);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "CUDA error in moveKernel: %s\n", hipGetErrorString(err));
	}

	hipDeviceSynchronize();

	constexpr float noise_low = -0.1f;
	constexpr float noise_high = 0.1f;
	int noise_seed = final_h_shift * 1000 + final_v_shift;

	addNoiseKernel<<<numBlocks, blockSize>>>(
	    result.getGpuData(),
	    noise_low,
	    noise_high,
	    noise_seed);

	hipError_t err2 = hipGetLastError();
	if (err2 != hipSuccess) {
		fprintf(stderr, "CUDA error in addNoiseKernel: %s\n", hipGetErrorString(err2));
	}

	hipDeviceSynchronize();
}
